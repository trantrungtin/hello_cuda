#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void hello_cuda() {
	printf("Hello Cuda blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d <> threadIdx.x=%d, threadIdx.y=%d, threadIdx.z=%d\n", 
		blockIdx.x, blockIdx.y, blockIdx.z,
		threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
	dim3 grid(2, 2); // number of blocks
	dim3 block(8, 2); // threads per block

	hello_cuda << <grid, block>> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}