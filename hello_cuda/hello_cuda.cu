#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void hello_cuda() {
	printf("Hello Cuda blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d <> threadIdx.x=%d, threadIdx.y=%d, threadIdx.z=%d\n", 
		blockIdx.x, blockIdx.y, blockIdx.z,
		threadIdx.x, threadIdx.y, threadIdx.z);
}

void sample1() {
	dim3 grid(2, 2); // number of blocks
	dim3 block(8, 2); // threads per block

	hello_cuda << <grid, block >> > ();
}

__global__ void unique_idx_calc_threadIdx(int* input) {
	int tid = threadIdx.x;
	printf("threadIdx: %d, value: %d\n", tid, input[tid]);
}

__global__ void unique_gid_calculation(int* input) {
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x=%d, threadIdx.x=%d, gid=%d, value=%d\n", blockIdx.x, threadIdx.x, gid, input[gid]);
}

void createCudaData(int* h_data, size_t size, int** d_data) {
	if (h_data == nullptr) {
		return;
	}
	hipMalloc((void**)d_data, size);
	hipMemcpy(*d_data, h_data, size, hipMemcpyHostToDevice);
}

void print(int* arr, size_t size) {
	size_t cnt = size / sizeof(int);
	for (int i = 0; i < cnt; i++) {
		printf("%d,", arr[i]);
	}
	printf("\n\n");
}

void sample2() {
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33 };
	print(h_data, sizeof(h_data));

	int* d_data;
	createCudaData(h_data, sizeof(h_data), &d_data);

	dim3 block(4);
	dim3 grid(2);

	//unique_idx_calc_threadIdx << <grid, block >> > (d_data);
	unique_gid_calculation << <grid, block >> > (d_data);
}

__global__ void unique_gid_calculation_2d(int* input) {
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = gridDim.x * blockDim.x * blockIdx.y;
	int gid = row_offset + block_offset + tid;
	printf("blockIdx.x=%d, threadIdx.x=%d, gid=%d, value=%d\n", 
		blockIdx.x, threadIdx.x, gid, input[gid]);
}

void sample3() {
	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 22, 43, 56, 4, 76, 81, 94, 32 };
	print(h_data, sizeof(h_data));
	int* d_data;
	createCudaData(h_data, sizeof(h_data), &d_data);
	dim3 block(4);
	dim3 grid(2, 2);
	unique_gid_calculation_2d << <grid, block >> > (d_data);
}

int main() {
	//sample1();
	//sample2();
	sample3();

	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}