#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void hello_cuda() {
	printf("Hello Cuda x=%d, y=%d, z=%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {
	dim3 grid(2, 2); // number of blocks
	dim3 block(8, 2); // threads per block

	hello_cuda << <grid, block>> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}