#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void hello_cuda() {
	printf("Hello Cuda\n");
}

int main() {
	dim3 grid(8); // number of blocks
	dim3 block(4); // threads per block

	hello_cuda << <grid, block>> > ();
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}